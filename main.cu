#include "hip/hip_runtime.h"
#include <cassert>
#include <iostream>
#include <sstream>
#include <stdexcept>
#include <cmath>
#include <cstdlib>

// #include <string>
#include <GL/glew.h>
#include <GLFW/glfw3.h>

#include "Program.h"
#include "Camera.h"

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#define BLOCKDIM_X 16
#define BLOCKDIM_Y 16

#define NK_INCLUDE_FIXED_TYPES
#define NK_INCLUDE_STANDARD_IO
#define NK_INCLUDE_DEFAULT_ALLOCATOR
#define NK_INCLUDE_VERTEX_BUFFER_OUTPUT
#define NK_INCLUDE_FONT_BAKING
#define NK_INCLUDE_DEFAULT_FONT
#define NK_IMPLEMENTATION
#define NK_GLFW_GL3_IMPLEMENTATION
#include "./nuklear/nuklear.h"
#include "./nuklear/nuklear_glfw_gl3.h"

#define MAX_VERTEX_BUFFER 512 * 1024
#define MAX_ELEMENT_BUFFER 128 * 1024

struct nk_context *ctx;

/****************
 *   parameter   *
 ****************/
float c = 2.99792458e8;
float freq = 1.0e9;
float lambda;
float resolution = 20.0;
float delta_x;
float delta_y;
float alpha = 0.5;
float delta_t;
float step;
float mu0;
float sigma0 = 0;
float epsilon0 = 8.854187e-12f;
float epsilonMax;
int M = 4;
int L = 12;
float r0 = -6;
float ecmax;
float Ez_max = 2.060459378159e-03f;
float Ez_min = -7.196258220476e-04f;
float Ez_range;
float Ez_yellow;
float Ez_green;
float Ez_lightblue;

// float pulse;
float T = 0.0;
bool flag = false;
int kt = 1;

float *h_Ez, *h_Hx, *h_Hy, *h_sigma_M, *h_epsilon_M, *h_mu_M;
float *h_ECX, *h_ECY;
float *h_CEZX, *h_CEZXL, *h_CHYX, *h_CHYXL, *h_CEZY, *h_CEZYL, *h_CHXY, *h_CHXYL;
float *h_EZX, *h_EZY, *h_HXY, *h_HYX;
float *h_CEZ, *h_CEZLX, *h_CEZLY, *h_CHXLY, *h_CHYLX;

float *d_Ez, *d_Hx, *d_Hy;
float *d_CEZX, *d_CEZXL, *d_CHYX, *d_CHYXL, *d_CEZY, *d_CEZYL, *d_CHXY, *d_CHXYL;
float *d_EZX, *d_EZY, *d_HXY, *d_HYX;
float *d_CEZ, *d_CEZLX, *d_CEZLY, *d_CHXLY, *d_CHYLX;

int wall_r;
int power_x, power_y;

//camera
float gScrollY = 0.0;





int SIZE_X=1080;
int SIZE_Y=1080;

/* int grid_x=256; */
/* int grid_y=256; */
int GRID_X=1080;
int GRID_Y=1080;

GLFWwindow* gWindow = NULL;
tdogl::Program* gProgram = NULL;
tdogl::Camera gCamera;

GLuint gVAO = 0;
GLuint gVBO = 0;
GLuint gTEX = 0;
GLuint gPBO = 0;
GLubyte *h_g_data;
GLubyte *d_g_data;
struct hipGraphicsResource *pbo_res;





void free_data(void);

void malloc_Initialdata(void);

void setInitialData(unsigned int width, unsigned int height);

void launchCPUKernel(GLubyte *g_data, float *Ez, float *Hx, float *Hy, float *CEZX, float *CEZXL, float *CHYX, float *CHYXL, float *CEZY, float *CEZYL, float *CHXY, float *CHXYL, float *EZX, float *EZY, float *HXY, float *HYX, float *CEZ, float *CEZLX, float *CEZLY, float *CHXLY, float *CHYLX, float step, unsigned int t, int L, unsigned int width, unsigned int height, float max, float min, float yellow, float green, float lightblue, int power_x, int power_y);

void launchGPUKernel(GLubyte *g_data, float *Ez, float *Hx, float *Hy, float *CEZX, float *CEZXL, float *CHYX, float *CHYXL, float *CEZY, float *CEZYL, float *CHXY, float *CHXYL, float *EZX, float *EZY, float *HXY, float *HYX, float *CEZ, float *CEZLX, float *CEZLY, float *CHXLY, float *CHYLX, float step, unsigned int t, int L, unsigned int width, unsigned int height, float max, float min, float yellow, float green, float lightblue, int power_x, int power_y, int R);

void h_FDTD2d_tm(GLubyte *g_data, float *Ez, float *Hx, float *Hy, float *CEZX, float *CEZXL, float *CHYX, float *CHYXL, float *CEZY, float *CEZYL, float *CHXY, float *CHXYL, float *EZX, float *EZY, float *HXY, float *HYX, float *CEZ, float *CEZLX, float *CEZLY, float *CHXLY, float *CHYLX, float step, unsigned int t, int L, unsigned int width, unsigned int height, float max, float min, float yellow, float green, float lightblue, int power_x, int power_y);

float h_clamp(float x, float a, float b);

__device__ float d_clamp(float x, float a, float b);

__global__ void d_FDTD2d_tm_H(GLubyte *g_data, float *Ez, float *Hx, float *Hy, float *CHYX, float *CHYXL, float *CHXY, float *CHXYL, float *HXY, float *HYX, float *CHXLY, float *CHYLX, int L, unsigned int width, unsigned int height, float max, float min, float yellow, float green, float lightblue);

__global__ void d_FDTD2d_tm_E(float *Ez, float *Hx, float *Hy, float *CEZX, float *CEZXL, float *CEZY, float *CEZYL, float *EZX, float *EZY, float *CEZ, float *CEZLX, float *CEZLY, float step, unsigned int t, int L, unsigned int width, unsigned int height, int power_x, int power_y);

float h_clamp(float x, float a, float b);


void RunGPUKernel(void);
void RunCPUKernel(void);

void InitPBO(GLuint *pbo, unsigned int size, struct hipGraphicsResource **pbo_res, unsigned int pbo_res_flags);

void InitTexData();

void LoadShaders();

void LoadTriangle();

void Render();

void OnError(int errorCode, const char* msg);

void AppMain();

float CalcFPS(GLFWwindow *gWindow, float theTimeInterval = 1.0, std::string theWindowTitle = "NONE");

void Update(float secondsElapsed);

void CameraInit();

void OnScroll(GLFWwindow *window, double deltaX, double deltaY);

void OnClick(GLFWwindow *window, int button, int action, int mods);

void PEC(GLubyte *h_g_data, float *ez, int X, int Y, int r);

void GUIRender(struct nk_context *ctx, int x, int y);




void GUIRender(struct nk_context *ctx, int x, int y)
{
  struct nk_panel layout;
  struct nk_rect bounds;
  const struct nk_input *in = &ctx->input;
  if (nk_begin(ctx, &layout, "Tools", nk_rect(50, 50, 300, 400),
        NK_WINDOW_BORDER|NK_WINDOW_MOVABLE|NK_WINDOW_SCALABLE|
        NK_WINDOW_MINIMIZABLE|NK_WINDOW_TITLE))
  {
    nk_layout_row_dynamic(ctx, 30, 2);
    bounds = nk_widget_bounds(ctx);
    if(nk_button_label(ctx, "Start/Stop"))
    {
      flag=!flag;
    }
    if(nk_button_label(ctx, "Restart"))
    {
      flag=false;
      kt=1;
      T=0.0;
      setInitialData(x, y);
      CameraInit();
      gCamera.setFieldOfView(50.0);
    }
    nk_layout_row_dynamic(ctx, 20, 2);
    nk_value_float_e(ctx, "Ez_Max", Ez_max);
    nk_value_float_e(ctx, "Ez_Min", Ez_min);

    if (nk_input_is_mouse_hovering_rect(in, bounds))
    {
      const struct nk_style *style;
      struct nk_panel layout;

      style=&ctx->style;

      if(nk_tooltip_begin(ctx, &layout, 130))
      {
        nk_layout_row_dynamic(ctx, style->font.height, 1);
        nk_text(ctx, "K -> MoveUp", 11, NK_TEXT_LEFT);
        nk_layout_row_dynamic(ctx, style->font.height, 1);
        nk_text(ctx, "J -> MoveDown", 13, NK_TEXT_LEFT);
        nk_layout_row_dynamic(ctx, style->font.height, 1);
        nk_text(ctx, "H -> MoveLeft", 13, NK_TEXT_LEFT);
        nk_layout_row_dynamic(ctx, style->font.height, 1);
        nk_text(ctx, "L -> MoveRight", 14, NK_TEXT_LEFT);
        nk_layout_row_dynamic(ctx, style->font.height, 1);
        nk_text(ctx, "Z -> ZoomIn", 11, NK_TEXT_LEFT);
        nk_layout_row_dynamic(ctx, style->font.height, 1);
        nk_text(ctx, "X -> ZoomOut", 12, NK_TEXT_LEFT);
        nk_layout_row_dynamic(ctx, style->font.height, 1);
        nk_text(ctx, "C -> ResetCamera", 16, NK_TEXT_LEFT);
        nk_tooltip_end(ctx);
      }
    }


    /* if (nk_tree_push(ctx, NK_TREE_NODE, "Sampling", NK_MINIMIZED)) */
    /* { */
    /*   int i; */
    /*   static int onoff=nk_false; */
    /*   static int pick=nk_false; */
    /*   nk_layout_row_dynamic(ctx, 20, 4); */
    /*   nk_checkbox_label(ctx, "On/Off", &onoff); */
    /*   nk_selectable_label(ctx, "Pick", NK_TEXT_LEFT, &pick); */
    /*   nk_value_int(ctx, "x", sampling[0]); */
    /*   nk_value_int(ctx, "y", ftoi(GRID_SIZE.y)-sampling[1]); */
    /*   if(onoff==nk_false) */
    /*     sampling_flag=false; */
    /*   else */
    /*     sampling_flag=true; */
    /*  */
    /*   if(pick==nk_false) */
    /*     sampling_pick=false; */
    /*   else */
    /*     sampling_pick=true; */
    /*  */
    /*   nk_layout_row_dynamic(ctx, 60, 1); */
    /*  */
    /*   if(nk_chart_begin(ctx, NK_CHART_LINES, 100, Ez_min, Ez_max)) */
    /*   { */
    /*     for(i=0;i<100;i++) */
    /*     { */
    /*       nk_chart_push(ctx, sampling_list[i]); */
    /*     } */
    /*     nk_chart_end(ctx); */
    /*   } */
    /*  */
    /*   nk_tree_pop(ctx); */
    /* } */

    if(nk_tree_push(ctx, NK_TREE_NODE, "Shape", NK_MINIMIZED))
    {
      nk_layout_row_begin(ctx, NK_STATIC, 20, 3);
      nk_layout_row_push(ctx, 20);
      nk_button_set_behavior(ctx, NK_BUTTON_REPEATER);
      if(nk_button_symbol(ctx, NK_SYMBOL_TRIANGLE_LEFT)){
        wall_r-=1;
        if(wall_r<=0){
          wall_r=0;
        }
      }

      nk_layout_row_push(ctx, 100);
      nk_value_float(ctx, "Width", wall_r);

      nk_layout_row_push(ctx, 20);
      nk_button_set_behavior(ctx, NK_BUTTON_REPEATER);
      if(nk_button_symbol(ctx, NK_SYMBOL_TRIANGLE_RIGHT)){
        wall_r+=1;
      }
      nk_layout_row_end(ctx);
      nk_tree_pop(ctx);
    }

  }
  nk_end(ctx);

}


void OnClick(GLFWwindow *window, int button, int action, int mods)
{
  if(button == GLFW_MOUSE_BUTTON_LEFT)
  {
    if(action == GLFW_PRESS)
    {
    }
  }
}

void OnScroll(GLFWwindow *window, double deltaX, double deltaY)
{
  gScrollY += deltaY;
}

void MouseInit(void)
{
  glfwSetCursorPos(gWindow, 0, 0);
  glfwSetScrollCallback(gWindow, OnScroll);
  glfwSetMouseButtonCallback(gWindow, OnClick);
}

void Update(float secondsElapsed)
{
  //keyboard
  const float moveSpeed = 0.5;
  if(glfwGetKey(gWindow, GLFW_KEY_K))
  {
    gCamera.offsetPosition(secondsElapsed * moveSpeed * glm::vec3(0, 1, 0));
  }else if(glfwGetKey(gWindow, GLFW_KEY_J))
  {
    gCamera.offsetPosition(secondsElapsed * moveSpeed * -glm::vec3(0, 1, 0));
  }

  if(glfwGetKey(gWindow, GLFW_KEY_H))
  {
    gCamera.offsetPosition(secondsElapsed * moveSpeed * -gCamera.right());
  }else if(glfwGetKey(gWindow, GLFW_KEY_L))
  {
    gCamera.offsetPosition(secondsElapsed * moveSpeed * gCamera.right());
  }

  if(glfwGetKey(gWindow, GLFW_KEY_Z))
  {
    gCamera.offsetPosition(secondsElapsed * moveSpeed * -gCamera.forward());
  }else if(glfwGetKey(gWindow, GLFW_KEY_X))
  {
    gCamera.offsetPosition(secondsElapsed * moveSpeed * gCamera.forward());
  }

  if(glfwGetKey(gWindow, GLFW_KEY_C))
  {
    CameraInit();
    gCamera.setFieldOfView(50.0);
    flag=true;
  }


  if(glfwGetKey(gWindow, GLFW_KEY_O))
  {
    wall_r-=1;
  }

  if(glfwGetKey(gWindow, GLFW_KEY_P))
  {
    wall_r+=1;
  }

  //mouse
  const float zoomSensitivity = -0.2f;
  float fieldOfView = gCamera.fieldOfView() + zoomSensitivity * (float)gScrollY;
  if(fieldOfView < 5.0f)
    fieldOfView = 5.0f;
  if(fieldOfView > 130.0f)
    fieldOfView = 130.0f;
  gCamera.setFieldOfView(fieldOfView);
  gScrollY = 0;

}

void CameraInit()
{
  gCamera.setPosition(glm::vec3(0, 0, 2.1));
  gCamera.setViewportAspectRatio(SIZE_X / SIZE_Y);
}


float CalcFPS(GLFWwindow *gWindow, float theTimeInterval, std::string theWindowTitle)
{
  static float t0Value = glfwGetTime();
  static int fpsFrameCount = 0;
  static float fps = 0.0;

  float currentTime = glfwGetTime();

  if(theTimeInterval < 0.1)
    theTimeInterval = 0.1;
  if(theTimeInterval > 10.0)
    theTimeInterval = 10.0;

  if((currentTime - t0Value) > theTimeInterval)
  {
    fps = (float)fpsFrameCount / (currentTime - t0Value);

    if(theWindowTitle != "NONE")
    {
      std::ostringstream stream;
      stream << fps;
      std::string fpsString = stream.str();

      theWindowTitle += " | FPS: " + fpsString;

      const char *pszConstString = theWindowTitle.c_str();
      glfwSetWindowTitle(gWindow, pszConstString);
    }else{
      std::cout << "FPS: " << fps << std::endl;
    }

    fpsFrameCount = 0;
    t0Value = glfwGetTime();
  }else{
    fpsFrameCount++;
  }
  return fps;
}

__device__ float d_clamp(float x, float a, float b)
{
  if (x < a){
    x = a;
  }
  if (x > b){
    x = b;
  }
  return x;
}


void PEC(GLubyte *h_g_data, float *ez, int X, int Y, int r){
  int index;
  /* for(int i=0;i<X;i++){ */
  /*   for(int j=0;j<Y/2-r/2;j++){ */
  /*     index = GRID_Y * j + i; */

  for(int i=0;i<X;i++){
    for(int j=0;j<Y/2-r/2;j++){
      index = GRID_Y * j + i;
      ez[index] = 0.0;
      h_g_data[index * 3 + 0]=(GLubyte)0;
      h_g_data[index * 3 + 1]=(GLubyte)0;
      h_g_data[index * 3 + 2]=(GLubyte)0;
    }
  }

  for(int i=0;i<X/2-r/2;i++){
    for(int j=Y/2+r/2;j<Y;j++){
      index = GRID_Y * j + i;
      ez[index] = 0.0;
      h_g_data[index * 3 + 0]=(GLubyte)0;
      h_g_data[index * 3 + 1]=(GLubyte)0;
      h_g_data[index * 3 + 2]=(GLubyte)0;
    }
  }

  for(int i=X/2+r/2;i<X;i++){
    for(int j=Y/2-r/2;j<Y;j++){
      index = GRID_Y * j + i;
      ez[index] = 0.0;
      h_g_data[index * 3 + 0]=(GLubyte)0;
      h_g_data[index * 3 + 1]=(GLubyte)0;
      h_g_data[index * 3 + 2]=(GLubyte)0;
    }
  }

  int j = Y/2-r/2;
  for(int i=X/2-r/2;i<X/2+r/2;i++){
    for(int k=Y/2-r/2;k<=j;k++){
      index = GRID_Y * k + i;
      ez[index] = 0.0;
      h_g_data[index * 3 + 0]=(GLubyte)0;
      h_g_data[index * 3 + 1]=(GLubyte)0;
      h_g_data[index * 3 + 2]=(GLubyte)0;
    }
    j+=(int)(sqrt(2));
  }

}

__device__ void GPU_PEC(GLubyte *h_g_data, float *ez, int X, int Y, int r){
  int i, j, index;
  i = blockDim.x * blockIdx.x + threadIdx.x;
  j = blockDim.y * blockIdx.y + threadIdx.y;
  int sqr = rsqrt(2.0);

  if(i<Y && j<X/2-r/2){
    index = Y * j + i;
    ez[index] = 0.0;
    h_g_data[index * 3 + 0]=(GLubyte)0;
    h_g_data[index * 3 + 1]=(GLubyte)0;
    h_g_data[index * 3 + 2]=(GLubyte)0;
  }
  if(i<Y/2-r/2 && j>= X/2+r/2 && j<X){
    index = Y * j + i;
    ez[index] = 0.0;
    h_g_data[index * 3 + 0]=(GLubyte)0;
    h_g_data[index * 3 + 1]=(GLubyte)0;
    h_g_data[index * 3 + 2]=(GLubyte)0;
  }
  if(i>=Y/2+r/2 && i<Y && j>=X/2-r/2 && j<X){
    index = Y * j + i;
    ez[index] = 0.0;
    h_g_data[index * 3 + 0]=(GLubyte)0;
    h_g_data[index * 3 + 1]=(GLubyte)0;
    h_g_data[index * 3 + 2]=(GLubyte)0;
  }

  if(i>=j){
    index = Y * j + i;
    ez[index] = 0.0;
    h_g_data[index * 3 + 0]=(GLubyte)0;
    h_g_data[index * 3 + 1]=(GLubyte)0;
    h_g_data[index * 3 + 2]=(GLubyte)0;
  }
}

__global__ void d_FDTD2d_tm_H(GLubyte *g_data, float *Ez, float *Hx, float *Hy, float *CHYX, float *CHYXL, float *CHXY, float *CHXYL, float *HXY, float *HYX, float *CHXLY, float *CHYLX, int L, unsigned int width, unsigned int height, float max, float min, float yellow, float green, float lightblue, int R)
{
  unsigned int i, j, index;
  float v;

  i = blockDim.x * blockIdx.x + threadIdx.x;
  j = blockDim.y * blockIdx.y + threadIdx.y;
  index = width * j + i;

  /*** range check ***/
  if ((i >= height) || (j >= width)){
    return;
  }

  /***create graphic data***/
  v = Ez[index];
  v = d_clamp(v, min, max);

  if(v > yellow) {
    g_data[index*3] = (GLubyte)255;
    g_data[index*3+1] = (GLubyte)(255-(v-yellow)/(max-yellow)*255);
    g_data[index*3+2] = (GLubyte)0;
  }else if(v > green){
    g_data[index*3] = (GLubyte)((v-green)/(yellow-green)*255);
    g_data[index*3+1] = (GLubyte)255;
    g_data[index*3+2] = (GLubyte)0;
  }else if(v > lightblue) {
    g_data[index*3] = (GLubyte)0;
    g_data[index*3+1] = (GLubyte)255;
    g_data[index*3+2] = (GLubyte)(255-(v-lightblue)/(green-lightblue)*255);
  }else{
    g_data[index*3] = (GLubyte)0;
    g_data[index*3+1] = (GLubyte)((v-min)/(lightblue-min)*255);
    g_data[index*3+2] = (GLubyte)255;
  }

  /*** range check (Hx)***/
  if ((i >= height-1) || (j >= width-1)){
    return;
  }
  //Hx
  Hx[index] = Hx[index] - (CHXLY[index]*(Ez[index+width]-Ez[index]));

  //Hx for PML
  if(i<L || i>width-L-1 || j<L || j>height-L-1){
    HXY[index]=CHXY[index]*HXY[index]-CHXYL[index]*(Ez[index+width]-Ez[index]);
    Hx[index]=HXY[index];
  }

  /*** range check (Hy)***/
  if (i == 0 || j == 0){
    return;
  }
  // Hy
  Hy[index] = Hy[index] + (CHYLX[index]*(Ez[index+1]-Ez[index]));

  //Hy for PML
  if(i<L || i>width-L-1 || j<L || j>height-L-1){
    HYX[index]=CHYX[index]*HYX[index]+CHYXL[index]*(Ez[index+1]-Ez[index]);
    Hy[index]=HYX[index];
  }
  
  GPU_PEC(g_data, Ez, width, height, R);
}

__global__ void d_FDTD2d_tm_E(float *Ez, float *Hx, float *Hy, float *CEZX, float *CEZXL, float *CEZY, float *CEZYL, float *EZX, float *EZY, float *CEZ, float *CEZLX, float *CEZLY, float step, unsigned int t, int L, unsigned int width, unsigned int height, int power_x, int power_y)
{
  unsigned int i, j, index;
  float pulse;

  i = blockDim.x * blockIdx.x + threadIdx.x;
  j = blockDim.y * blockIdx.y + threadIdx.y;
  index = width * j + i;

  /*** range check ***/
  if ((i >= height-1) || (j >= width-1) || i == 0 || j == 0){
    return;
  }

  pulse  =  sin((((t - 1)%(int)step)+1)*2.0*M_PI/step);

  //Ez
  if(i==power_x && j==power_y){
    Ez[index] = 1.0/376.7 * pulse;
  }else{
    Ez[index] = CEZ[index] * Ez[index] + CEZLX[index] * (Hy[index]-Hy[index-1]) - CEZLY[index] * (Hx[index]-Hx[index-width]);
  }

  if(i<L || (i>width-L-1) || j<L || (j>height-L-1)){
    EZX[index]=CEZX[index] * EZX[index] + CEZXL[index] * (Hy[index] - Hy[index-1]);
    EZY[index]=CEZY[index] * EZY[index] - CEZYL[index] * (Hx[index] - Hx[index-width]);
    Ez[index]=EZX[index]+EZY[index];
  }	
}

void launchGPUKernel(GLubyte *g_data, float *Ez, float *Hx, float *Hy, float *CEZX, float *CEZXL, float *CHYX, float *CHYXL, float *CEZY, float *CEZYL, float *CHXY, float *CHXYL, float *EZX, float *EZY, float *HXY, float *HYX, float *CEZ, float *CEZLX, float *CEZLY, float *CHXLY, float *CHYLX, float step, unsigned int t, int L, unsigned int width, unsigned int height, float max, float min, float yellow, float green, float lightblue, int power_x, int power_y, int R)
{
  dim3 grid(width / BLOCKDIM_X + 1, height / BLOCKDIM_Y + 1);
  dim3 block(BLOCKDIM_X, BLOCKDIM_Y, 1);


  d_FDTD2d_tm_E <<< grid, block >>> (Ez, Hx, Hy, CEZX, CEZXL, CEZY, CEZYL, EZX, EZY, CEZ, CEZLX, CEZLY, step, t, L, width, height, power_x, power_y);
  d_FDTD2d_tm_H <<< grid, block >>> (g_data, Ez, Hx, Hy, CHYX, CHYXL, CHXY, CHXYL, HXY, HYX, CHXLY, CHYLX, L, width, height, max, min, yellow, green, lightblue, R);
}


float h_clamp(float x, float a, float b)
{
  if (x < a)
    x = a;
  if (x > b)
    x = b;
  return x;
}

void h_FDTD2d_tm(GLubyte *g_data, float *Ez, float *Hx, float *Hy, float *CEZX, float *CEZXL, float *CHYX, float *CHYXL, float *CEZY, float *CEZYL, float *CHXY, float *CHXYL, float *EZX, float *EZY, float *HXY, float *HYX, float *CEZ, float *CEZLX, float *CEZLY, float *CHXLY, float *CHYLX, float step, unsigned int t, int L, unsigned int width, unsigned int height, float max, float min, float yellow, float green, float lightblue, int power_x, int power_y)
{
  unsigned int i, j, index;
  float pulse;
  pulse  =  sin((((t - 1)%(int)step)+1)*2.0*M_PI/step);

  //Ez
  for(j = 1; j < height-1; j++){
    for(i = 1; i < width-1; i++){
      index = width * j + i;
      if(i==power_x && j==power_y){
        Ez[index] = 1.0/376.7 * pulse;
      }else{
        Ez[index] = CEZ[index] * Ez[index] + CEZLX[index] * (Hy[index]-Hy[index-1]) - CEZLY[index] * (Hx[index]-Hx[index-width]);
      }
    }
  }

  /* Ez for PML */
  for(j = 1; j<height - 1; j++){
    for(i = 1; i<width - 1; i++){
      index = width * j + i;
      if(i<L || (i>width-L-1) || j<L || (j>height-L-1)){
        EZX[index]=CEZX[index] * EZX[index] + CEZXL[index] * (Hy[index] - Hy[index-1]);
        EZY[index]=CEZY[index] * EZY[index] - CEZYL[index] * (Hx[index] - Hx[index-width]);
        Ez[index]=EZX[index]+EZY[index];
      }
    }
  }

  // T=T+delta_t/2;

  //Hx
  for(j = 0; j<height - 1; j++){
    for(i = 1; i<width - 1; i++){
      index = width * j + i;
      Hx[index] = Hx[index] - (CHXLY[index]*(Ez[index+width]-Ez[index]));
    }
  }

  /* //Hx for PML*/
  for(j = 0; j<height - 1; j++){
    for(i = 1; i<width - 1; i++){
      index = width * j + i;
      if(i<L || i>width-L-1 || j<L || j>height-L-1){
        HXY[index]=CHXY[index]*HXY[index]-CHXYL[index]*(Ez[index+width]-Ez[index]);
        Hx[index]=HXY[index];
      }
    }
  }

  //Hy
  for(j = 1; j<height - 1; j++){
    for(i = 0; i<width - 1; i++){
      index = width * j + i;
      Hy[index] = Hy[index] + (CHYLX[index]*(Ez[index+1]-Ez[index]));
    }
  }

  //Hy for PML
  for(j = 1; j<height - 1; j++){
    for(i = 0; i<width - 1; i++){
      index = width * j + i;
      if(i<L || i>width-L-1 || j<L || j>height-L-1){
        HYX[index]=CHYX[index]*HYX[index]+CHYXL[index]*(Ez[index+1]-Ez[index]);
        Hy[index]=HYX[index];
      }
    }
  }

  /* Blank_Wall(g_data, Ez, wall_r, width, height); */

  // T=T+delta_t/2;


  /***create graphic data***/
  float v;
  for(j=0; j<height; j++){
    for(i=0; i<width; i++){
      index = width * j + i;
      v = Ez[index];
      v = h_clamp(v, min, max);

      if(v > yellow) {
        g_data[index*3] = (GLubyte)255;
        g_data[index*3+1] = (GLubyte)(255-(v-yellow)/(max-yellow)*255);
        g_data[index*3+2] = (GLubyte)0;
      }else if(v > Ez_green){
        g_data[index*3] = (GLubyte)((v-green)/(yellow-green)*255);
        g_data[index*3+1] = (GLubyte)255;
        g_data[index*3+2] = (GLubyte)0;
      }else if(v > lightblue) {
        g_data[index*3] = (GLubyte)0;
        g_data[index*3+1] = (GLubyte)255;
        g_data[index*3+2] = (GLubyte)(255-(v-lightblue)/(green-lightblue)*255);
      }else{
        g_data[index*3] = (GLubyte)0;
        g_data[index*3+1] = (GLubyte)((v-min)/(lightblue-min)*255);
        g_data[index*3+2] = (GLubyte)255;
      }
    }
  }

  PEC(g_data, Ez, width, height, wall_r);
}

void launchCPUKernel(GLubyte *g_data, float *Ez, float *Hx, float *Hy, float *CEZX, float *CEZXL, float *CHYX, float *CHYXL, float *CEZY, float *CEZYL, float *CHXY, float *CHXYL, float *EZX, float *EZY, float *HXY, float *HYX, float *CEZ, float *CEZLX, float *CEZLY, float *CHXLY, float *CHYLX, float step, unsigned int t, int L, unsigned int width, unsigned int height, float max, float min, float yellow, float green, float lightblue, int power_x, int power_y)
{
  h_FDTD2d_tm(g_data, Ez, Hx, Hy, CEZX, CEZXL, CHYX, CHYXL, CEZY, CEZYL, CHXY, CHXYL, EZX, EZY, HXY, HYX, CEZ, CEZLX, CEZLY, CHXLY, CHYLX, step, t, L, width, height, max, min, yellow, green, lightblue, power_x, power_y);

}


void setInitialData(unsigned int width, unsigned int height)
{
  lambda = c / freq;
  delta_x = lambda / resolution;
  delta_y = lambda / resolution;	
  delta_t = (1.0 / (sqrt(pow((1 / delta_x), 2.0)+pow((1 / delta_y), 2.0))))*(1.0 / c)*alpha;
  step = 1.0 / freq / delta_t;
  mu0 = 1.0e-7f * 4.0 * M_PI;
  ecmax = -(M+1)*epsilon0*c / (2.0*L*delta_x)*r0;
  Ez_range = Ez_max-Ez_min; // 2.7800852e-03f 
  Ez_yellow = Ez_range*0.75f+Ez_min;
  Ez_green = Ez_range*0.50f+Ez_min;
  Ez_lightblue = Ez_range*0.25f+Ez_min;
  wall_r = lambda / 2 / delta_x + 10;

  power_x = 12;
  power_y = GRID_Y/2 - 1;

  int i, j, index;
  float Z, ZZ;
  for(j = 0; j<GRID_Y; j++){
    for(i = 0; i<GRID_X; i++){
      index = GRID_X * j + i;
      h_mu_M[index]  =  mu0;
      h_epsilon_M[index] = epsilon0;
      h_sigma_M[index] = sigma0;
    }
  }

  for(j = 0; j<GRID_Y; j++){
    for(i = 0;i<GRID_X; i++){
      index = GRID_X * j + i;
      h_Ez[index] = 0.0;
      h_Hx[index] = 0.0;
      h_Hy[index] = 0.0;
      h_CEZX[index] = 0.0;
      h_CEZXL[index] = 0.0;
      h_CHYX[index] = 0.0;
      h_CHYXL[index] = 0.0;
      h_CEZY[index] = 0.0;
      h_CEZYL[index] = 0.0;
      h_CHXY[index] = 0.0;
      h_CHXYL[index] = 0.0;

      h_CEZ[index] = 0.0;
      h_CEZLX[index]=0.0;
      h_CEZLY[index]=0.0;
      h_CHXLY[index]=0.0;
      h_CHYLX[index]=0.0;
    }
  }

  for(i=0;i<GRID_X;i++){
    h_ECX[i]=0.0;
  }

  for(j=0;j<GRID_X;j++){
    h_ECY[j]=0.0;
  }

  for(i=0;i<L;i++){
    h_ECX[i] = ecmax * pow((L-i+0.5)/L,M);
    h_ECX[GRID_X-i-1] = h_ECX[i];
    h_ECY[i] = h_ECX[i];
    h_ECY[GRID_Y-i-1] = h_ECX[i];
  }

  //PML init
  for(i=0;i<GRID_X;i++){
    for(j=0;j<GRID_Y;j++){
      index = GRID_X * j + i;
      Z = (h_ECX[i] * delta_t)/(2.0*h_epsilon_M[index]);
      h_CEZX[index]=(1-Z)/(1+Z);
      h_CEZXL[index]=(delta_t/h_epsilon_M[index])/(1+Z)*(1.0/delta_x);
      h_CHYX[index]=(1-Z)/(1+Z);
      h_CHYXL[index]=(delta_t/h_mu_M[index])*(1.0/delta_x);
      Z = (h_ECY[j]*delta_t)/(2.0*h_epsilon_M[index]);
      h_CEZY[index]=(1-Z)/(1+Z);
      h_CEZYL[index]=(delta_t/h_epsilon_M[index])/(1+Z)*(1.0/delta_y);
      h_CHXY[index]=(1-Z)/(1+Z);
      h_CHXYL[index]=(delta_t/h_mu_M[index])*(1.0/delta_y);
    }
  }

  //FDTD init
  for(i=0;i<GRID_X;i++){
    for(j=0;j<GRID_Y;j++){
      index = GRID_X * j + i;
      ZZ = (h_sigma_M[index] * delta_t)/(2.0*h_epsilon_M[index]);
      h_CEZ[index]=(1-ZZ)/(1+ZZ);
      h_CEZLX[index]=(delta_t/h_epsilon_M[index])/(1+ZZ)*(1.0/delta_x);
      h_CEZLY[index]=(delta_t/h_epsilon_M[index])/(1+ZZ)*(1.0/delta_y);
      h_CHXLY[index]=delta_t/h_mu_M[index]*(1.0/delta_y);
      h_CHYLX[index]=delta_t/h_mu_M[index]*(1.0/delta_x);
    }
  }

  for(i=0;i<GRID_X;i++){
    for(j=0;j<GRID_Y;j++){
      index = GRID_X * j + i;
      h_g_data[index*3] = (GLubyte)0;
      h_g_data[index*3+1] = (GLubyte)0;
      h_g_data[index*3+2] = (GLubyte)0;	
    }
  }

  hipMemcpy(d_Ez, h_Ez, sizeof(float) * GRID_Y * GRID_X, hipMemcpyHostToDevice);
  hipMemcpy(d_Hx, h_Hx, sizeof(float) * GRID_Y * GRID_X, hipMemcpyHostToDevice);
  hipMemcpy(d_Hy, h_Hy, sizeof(float) * GRID_Y * GRID_X, hipMemcpyHostToDevice);
  hipMemcpy(d_CEZX, h_CEZX, sizeof(float) * GRID_Y * GRID_X, hipMemcpyHostToDevice);
  hipMemcpy(d_CEZXL, h_CEZXL, sizeof(float) * GRID_Y * GRID_X, hipMemcpyHostToDevice);
  hipMemcpy(d_CHYX, h_CHYX, sizeof(float) * GRID_Y * GRID_X, hipMemcpyHostToDevice);
  hipMemcpy(d_CHYXL, h_CHYXL, sizeof(float) * GRID_Y * GRID_X, hipMemcpyHostToDevice);
  hipMemcpy(d_CEZY, h_CEZY, sizeof(float) * GRID_Y * GRID_X, hipMemcpyHostToDevice);
  hipMemcpy(d_CEZYL, h_CEZYL, sizeof(float) * GRID_Y * GRID_X, hipMemcpyHostToDevice);
  hipMemcpy(d_CHXY, h_CHXY, sizeof(float) * GRID_Y * GRID_X, hipMemcpyHostToDevice);
  hipMemcpy(d_CHXYL, h_CHXYL, sizeof(float) * GRID_Y * GRID_X, hipMemcpyHostToDevice);
  hipMemcpy(d_EZX, h_EZX, sizeof(float) * GRID_Y * GRID_X, hipMemcpyHostToDevice);
  hipMemcpy(d_EZY, h_EZY, sizeof(float) * GRID_Y * GRID_X, hipMemcpyHostToDevice);
  hipMemcpy(d_HXY, h_HXY, sizeof(float) * GRID_Y * GRID_X, hipMemcpyHostToDevice);
  hipMemcpy(d_HYX, h_HYX, sizeof(float) * GRID_Y * GRID_X, hipMemcpyHostToDevice);
  hipMemcpy(d_CEZ, h_CEZ, sizeof(float) * GRID_Y * GRID_X, hipMemcpyHostToDevice);
  hipMemcpy(d_CEZLX, h_CEZLX, sizeof(float) * GRID_Y * GRID_X, hipMemcpyHostToDevice);
  hipMemcpy(d_CEZLY, h_CEZLY, sizeof(float) * GRID_Y * GRID_X, hipMemcpyHostToDevice);
  hipMemcpy(d_CHXLY, h_CHXLY, sizeof(float) * GRID_Y * GRID_X, hipMemcpyHostToDevice);
  hipMemcpy(d_CHYLX, h_CHYLX, sizeof(float) * GRID_Y * GRID_X, hipMemcpyHostToDevice);
}


void malloc_Initialdata(void)
{

  h_g_data = (GLubyte *)malloc(sizeof(GLubyte) * GRID_X * GRID_Y * 3);

  h_Ez  = (float *)malloc(sizeof(float) * GRID_Y * GRID_X);
  h_Hx  = (float *)malloc(sizeof(float) * GRID_Y * GRID_X);
  h_Hy  = (float *)malloc(sizeof(float) * GRID_Y * GRID_X);
  hipMalloc((void**)&d_Ez, sizeof(float) *GRID_Y * GRID_X);
  hipMalloc((void**)&d_Hx, sizeof(float) *GRID_Y * GRID_X);
  hipMalloc((void**)&d_Hy, sizeof(float) *GRID_Y * GRID_X);

  h_sigma_M  = (float *)malloc(sizeof(float) * GRID_Y * GRID_X);
  h_epsilon_M = (float *)malloc(sizeof(float) * GRID_Y * GRID_X);
  h_mu_M = (float *)malloc(sizeof(float) * GRID_Y * GRID_X);

  h_ECX = (float *)malloc(sizeof(float) * GRID_X);
  h_ECY = (float *)malloc(sizeof(float) * GRID_Y);

  h_CEZX  = (float *)malloc(sizeof(float) * GRID_Y * GRID_X);
  h_CEZXL = (float *)malloc(sizeof(float) * GRID_Y * GRID_X);
  h_CHYX  = (float *)malloc(sizeof(float) * GRID_Y * GRID_X);
  h_CHYXL = (float *)malloc(sizeof(float) * GRID_Y * GRID_X);
  h_CEZY  = (float *)malloc(sizeof(float) * GRID_Y * GRID_X);
  h_CEZYL = (float *)malloc(sizeof(float) * GRID_Y * GRID_X);
  h_CHXY  = (float *)malloc(sizeof(float) * GRID_Y * GRID_X);
  h_CHXYL = (float *)malloc(sizeof(float) * GRID_Y * GRID_X);
  hipMalloc((void**)&d_CEZX, sizeof(float) *GRID_Y * GRID_X);
  hipMalloc((void**)&d_CEZXL, sizeof(float) *GRID_Y * GRID_X);
  hipMalloc((void**)&d_CHYX, sizeof(float) *GRID_Y * GRID_X);
  hipMalloc((void**)&d_CHYXL, sizeof(float) *GRID_Y * GRID_X);
  hipMalloc((void**)&d_CEZY, sizeof(float) *GRID_Y * GRID_X);
  hipMalloc((void**)&d_CEZYL, sizeof(float) *GRID_Y * GRID_X);
  hipMalloc((void**)&d_CHXY, sizeof(float) *GRID_Y * GRID_X);
  hipMalloc((void**)&d_CHXYL, sizeof(float) *GRID_Y * GRID_X);

  h_EZX = (float *)malloc(sizeof(float) * GRID_Y * GRID_X);
  h_EZY  = (float *)malloc(sizeof(float) * GRID_Y * GRID_X);
  h_HXY = (float *)malloc(sizeof(float) * GRID_Y * GRID_X);
  h_HYX = (float *)malloc(sizeof(float) * GRID_Y * GRID_X);
  hipMalloc((void**)&d_EZX, sizeof(float) *GRID_Y * GRID_X);
  hipMalloc((void**)&d_EZY, sizeof(float) *GRID_Y * GRID_X);
  hipMalloc((void**)&d_HXY, sizeof(float) *GRID_Y * GRID_X);
  hipMalloc((void**)&d_HYX, sizeof(float) *GRID_Y * GRID_X);

  h_CEZ  = (float *)malloc(sizeof(float) * GRID_Y * GRID_X);
  h_CEZLX = (float *)malloc(sizeof(float) * GRID_Y * GRID_X);
  h_CEZLY  = (float *)malloc(sizeof(float) * GRID_Y * GRID_X);
  h_CHXLY = (float *)malloc(sizeof(float) * GRID_Y * GRID_X);
  h_CHYLX  = (float *)malloc(sizeof(float) * GRID_Y * GRID_X);
  hipMalloc((void**)&d_CEZ, sizeof(float) *GRID_Y * GRID_X);
  hipMalloc((void**)&d_CEZLX, sizeof(float) *GRID_Y * GRID_X);
  hipMalloc((void**)&d_CEZLY, sizeof(float) *GRID_Y * GRID_X);
  hipMalloc((void**)&d_CHXLY, sizeof(float) *GRID_Y * GRID_X);
  hipMalloc((void**)&d_CHYLX, sizeof(float) *GRID_Y * GRID_X);
}

void free_data(void)
{
  free(h_Ez);
  free(h_Hx);
  free(h_Hy);
  free(h_ECX);
  free(h_ECY);
  free(h_CEZX);
  free(h_CEZXL);
  free(h_CHYX);
  free(h_CHYXL);
  free(h_CEZY);
  free(h_CEZYL);
  free(h_CHXY);
  free(h_CHXYL);
  free(h_EZX);
  free(h_EZY);
  free(h_HXY);
  free(h_HYX);
  free(h_CEZ);
  free(h_CEZLX);
  free(h_CEZLY);
  free(h_CHXLY);
  free(h_CHYLX);
  free(h_sigma_M);
  free(h_epsilon_M);
  free(h_mu_M);

  hipFree(d_Ez);
  hipFree(d_Hx);
  hipFree(d_Hy);
  hipFree(d_CEZX);
  hipFree(d_CEZXL);
  hipFree(d_CHYX);
  hipFree(d_CHYXL);
  hipFree(d_CEZY);
  hipFree(d_CEZYL);
  hipFree(d_CHXY);
  hipFree(d_CHXYL);
  hipFree(d_EZX);
  hipFree(d_EZY);
  hipFree(d_HXY);
  hipFree(d_HYX);
  hipFree(d_CEZ);
  hipFree(d_CEZLX);
  hipFree(d_CEZLY);
  hipFree(d_CHXLY);
  hipFree(d_CHYLX);
}














void RunGPUKernel(void){
  hipGraphicsMapResources(1, &pbo_res, 0);
  hipGraphicsResourceGetMappedPointer((void**)&d_g_data, NULL, pbo_res);

  if(!flag){
    launchGPUKernel(d_g_data, d_Ez, d_Hx, d_Hy, d_CEZX, d_CEZXL, d_CHYX, d_CHYXL, d_CEZY, d_CEZYL, d_CHXY, d_CHXYL, d_EZX, d_EZY, d_HXY, d_HYX, d_CEZ, d_CEZLX, d_CEZLY, d_CHXLY, d_CHYLX, step, kt, L, GRID_X, GRID_Y, Ez_max, Ez_min, Ez_yellow, Ez_green, Ez_lightblue, power_x, power_y, wall_r);
  }
  kt++;

  hipGraphicsUnmapResources(1, &pbo_res, 0);
  glBindBuffer(GL_PIXEL_UNPACK_BUFFER, gPBO);
  glBindTexture(GL_TEXTURE_2D, gTEX);
  glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, GRID_X, GRID_Y, GL_RGB, GL_UNSIGNED_BYTE, NULL);
  glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
}

void RunCPUKernel(void){

  if(!flag){
    launchCPUKernel(h_g_data, h_Ez, h_Hx, h_Hy, h_CEZX, h_CEZXL, h_CHYX, h_CHYXL, h_CEZY, h_CEZYL, h_CHXY, h_CHXYL, h_EZX, h_EZY, h_HXY, h_HYX, h_CEZ, h_CEZLX, h_CEZLY, h_CHXLY, h_CHYLX, step, kt, L, GRID_X, GRID_Y, Ez_max, Ez_min, Ez_yellow, Ez_green, Ez_lightblue, power_x, power_y);
  }


  kt++;

  glBindTexture(GL_TEXTURE_2D, gTEX);
  glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, GRID_X, GRID_Y, GL_RGB, GL_UNSIGNED_BYTE, h_g_data);
}
void InitPBO(GLuint *pbo, unsigned int size, struct hipGraphicsResource **pbo_res, unsigned int pbo_res_flags){
  glGenBuffers(1, pbo);
  glBindBuffer(GL_ARRAY_BUFFER, *pbo);
  glBufferData(GL_ARRAY_BUFFER, size, NULL, GL_DYNAMIC_DRAW);
  glBindBuffer(GL_ARRAY_BUFFER, 0);
  hipGraphicsGLRegisterBuffer(pbo_res, *pbo, pbo_res_flags);
}

void InitTexData(){
  int i, j;
  for(i=0;i<GRID_X;i++){
    for(j=0;j<GRID_Y;j++){
      int index = i*GRID_X+j;
      h_g_data[index*3+0] = (GLubyte)i;
      h_g_data[index*3+1] = (GLubyte)GRID_X-i;
      h_g_data[index*3+2] = (GLubyte)GRID_X-i;
    }
  }
  glGenTextures(1, &gTEX);
  glBindTexture(GL_TEXTURE_2D, gTEX);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
  glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, GRID_X, GRID_Y, 0, GL_RGB, GL_UNSIGNED_BYTE, NULL);
}

void LoadShaders(){
  std::vector<tdogl::Shader::Shader> shaders;
  shaders.push_back(tdogl::Shader::shaderFromFile("./vertex-shader.glsl", GL_VERTEX_SHADER));
  shaders.push_back(tdogl::Shader::shaderFromFile("./fragment-shader.glsl", GL_FRAGMENT_SHADER));
  gProgram = new tdogl::Program(shaders);
}

// loads a triangle into the VAO global
void LoadTriangle() {
  // make and bind the VAO
  glGenVertexArrays(1, &gVAO);
  glBindVertexArray(gVAO);

  // make and bind the VBO
  glGenBuffers(1, &gVBO);
  glBindBuffer(GL_ARRAY_BUFFER, gVBO);

  GLfloat vertexData[] = {
    1.0, -1.0, 0.0f, 1.0f, 0.0f,
    -1.0, -1.0, 0.0f, 0.0f, 0.0f,
    1.0,  1.0, 0.0f, 1.0f, 1.0f,

    1.0,  1.0, 0.0f, 1.0f, 1.0f,
    -1.0, -1.0, 0.0f, 0.0f, 0.0f,
    -1.0,  1.0, 0.0f, 0.0f, 1.0f

  };
  glBufferData(GL_ARRAY_BUFFER, sizeof(vertexData), vertexData, GL_STATIC_DRAW);

  // connect the xyz to the "vert" attribute of the vertex shader
  glEnableVertexAttribArray(gProgram->attrib("vert"));
  glVertexAttribPointer(gProgram->attrib("vert"), 3, GL_FLOAT, GL_FALSE, 5*sizeof(GLfloat), NULL);

  glEnableVertexAttribArray(gProgram->attrib("verTexCoord"));
  glVertexAttribPointer(gProgram->attrib("verTexCoord"), 2, GL_FLOAT, GL_FALSE, 5*sizeof(GLfloat), (const GLvoid*)(3*sizeof(GLfloat)));

  // unbind the VBO and VAO
  glBindBuffer(GL_ARRAY_BUFFER, 0);
  glBindVertexArray(0);
}


// draws a single frame
void Render() {
  // clear everything
  glClearColor(255, 255, 255, 1); // black
  glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

  // bind the program (the shaders)
  glUseProgram(gProgram->object());

  gProgram->setUniform("camera", gCamera.matrix());

  glActiveTexture(GL_TEXTURE0);
  glBindTexture(GL_TEXTURE_2D, gTEX);
  glUniform1f(gProgram->uniform("tex"), 0);

  // bind the VAO (the triangle)
  glBindVertexArray(gVAO);

  // draw the VAO
  glDrawArrays(GL_TRIANGLES, 0, 6);


  // unbind the VAO
  glBindVertexArray(0);

  glBindTexture(GL_TEXTURE_2D, 0);
  // unbind the program
  glUseProgram(0);

}

void OnError(int errorCode, const char* msg) {
  throw std::runtime_error(msg);
}

// the program starts here
void AppMain() {
  // initialise GLFW
  glfwSetErrorCallback(OnError);
  if(!glfwInit())
    throw std::runtime_error("glfwInit failed");

  // open a window with GLFW
  glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);
  glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
  glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
  glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 2);
  glfwWindowHint(GLFW_RESIZABLE, GL_FALSE);
  gWindow = glfwCreateWindow(SIZE_X, SIZE_Y, "Strting...", NULL, NULL);
  if(!gWindow)
    throw std::runtime_error("glfwCreateWindow failed. Can your hardware handle OpenGL 3.2?");

  // GLFW settings
  glfwMakeContextCurrent(gWindow);

  // initialise GLEW
  glewExperimental = GL_TRUE; //stops glew crashing on OSX :-/
  if(glewInit() != GLEW_OK)
    throw std::runtime_error("glewInit failed");

  // print out some info about the graphics drivers
  std::cout << "OpenGL version: " << glGetString(GL_VERSION) << std::endl;
  std::cout << "GLSL version: " << glGetString(GL_SHADING_LANGUAGE_VERSION) << std::endl;
  std::cout << "Vendor: " << glGetString(GL_VENDOR) << std::endl;
  std::cout << "Renderer: " << glGetString(GL_RENDERER) << std::endl;

  // make sure OpenGL version 3.2 API is available
  if(!GLEW_VERSION_3_2)
    throw std::runtime_error("OpenGL 3.2 API is not available.");







  malloc_Initialdata();

  setInitialData(GRID_X, GRID_Y);

  LoadShaders();



  InitPBO(&gPBO, sizeof(GLubyte)*GRID_X*GRID_Y*3, &pbo_res, hipGraphicsRegisterFlagsWriteDiscard);

  // load vertex and fragment shaders into opengl
  InitTexData();  

  // create buffer and fill it with the points of the triangle
  LoadTriangle();

  CameraInit();

  MouseInit();

  ctx = nk_glfw3_init(gWindow, NK_GLFW3_INSTALL_CALLBACKS);
  {
    struct nk_font_atlas *atlas;
    nk_glfw3_font_stash_begin(&atlas);
    nk_glfw3_font_stash_end();
  }


  float lastTime = glfwGetTime();
  // run while the window is open
  while(!glfwWindowShouldClose(gWindow)){
    // process pending events
    glfwPollEvents();

    nk_glfw3_new_frame();

    /* RunCPUKernel(); */
    RunGPUKernel();
    // draw one frame
    Render();

    CalcFPS(gWindow, 1.0, "GL3.2 FDTD2D_TM");

    float thisTime = glfwGetTime();
    Update((float)(thisTime - lastTime));

    
    GUIRender(ctx, GRID_X, GRID_Y);
    nk_glfw3_render(NK_ANTI_ALIASING_ON, MAX_VERTEX_BUFFER, MAX_ELEMENT_BUFFER);


    // swap the display buffers (displays what was just drawn)
    glfwSwapBuffers(gWindow);

    
    if(glfwGetKey(gWindow, GLFW_KEY_ESCAPE)){
      free_data();
      glDeleteTextures(1, &gTEX);
      hipGraphicsUnregisterResource(pbo_res);
      glDeleteBuffers(1, &gPBO);
      hipFree(d_g_data);
      free(h_g_data);
      hipDeviceReset();
      glfwSetWindowShouldClose(gWindow, GL_TRUE);
    }

  }
  // clean up and exit
  nk_glfw3_shutdown();
  glfwTerminate();
}

int main(int argc, char *argv[]) {
  try {
    AppMain();
  } catch (const std::exception& e){
    std::cerr << "ERROR: " << e.what() << std::endl;
    return EXIT_FAILURE;
  }

  return EXIT_SUCCESS;
}

